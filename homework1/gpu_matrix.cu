////////////////////////////////////////////////////////////////////////////////
//
// FILE:        gpu_matrix.cu
// DESCRIPTION: calculates matrix multiplication on gpu
// AUTHOR:      Dan Fabian
// DATE:        1/26/2020


#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>

using std::cout; using std::endl;
using namespace std::chrono;

// create matrices of size SIZExSIZE
const int SIZE = 128;

// kernel multiply func
__global__ void multiply(bool *a, bool *b, int *c)
{
	int xIdx = blockIdx.x, yIdx = threadIdx.x;
	for (int i = 0; i < SIZE; ++i)
		c[xIdx * SIZE + yIdx] += a[xIdx * SIZE + i] * b[i * SIZE + yIdx];	
}

int main()
{
	// create matrices
	bool a[SIZE][SIZE], b[SIZE][SIZE], *ad, *bd;
	int c[SIZE][SIZE], *cd;

	// create rng
	unsigned int seed = system_clock::now().time_since_epoch().count();
	std::default_random_engine generator(seed);
	std::uniform_int_distribution<int> dist(0, 1);

	// init matrices
	for (int i = 0; i < SIZE; ++i) for (int j = 0; j < SIZE; ++j)
	{
		a[i][j] = dist(generator);
		b[i][j] = dist(generator);
		c[i][j] = 0;
	}

	// memory size
	int boolSize = SIZE * SIZE * sizeof(bool);
	int intSize = SIZE * SIZE * sizeof(int);

	// allocate memory on device
	hipMalloc((void**)&ad, boolSize);
	hipMalloc((void**)&bd, boolSize);
	hipMalloc((void**)&cd, intSize);

	// copy a and b to device memory
	hipMemcpy(ad, a, boolSize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, boolSize, hipMemcpyHostToDevice);

	// call multiply func
	multiply<<<SIZE, SIZE>>>(ad, bd, cd);

	// copy device memory back to host
	hipMemcpy(c, cd, intSize, hipMemcpyDeviceToHost);

	// print out final matrix
	for (int i = 0; i < SIZE; ++i) 
	{
		for (int j = 0; j < SIZE; ++j)
			cout << c[i][j] << ' ';
		cout << endl;
	}

	// free all device memory
	hipFree(ad); hipFree(bd); hipFree(cd);

	return 0;
}
