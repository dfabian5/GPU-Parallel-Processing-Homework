////////////////////////////////////////////////////////////////////////////////
//
// FILE:        n_sample_moving_avg.cu
// DESCRIPTION: runs N Sample Moving Average Filtering algorithm on gpu
// AUTHOR:      Dan Fabian
// DATE:        2/16/2020


#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>

using std::cout; using std::endl; using std::cin;
using namespace std::chrono;

const int NUM_OF_VALS = 10000, N = 256, NUM_OF_AVG = NUM_OF_VALS - N + 1;

// kernal func
__global__ void movingAvg(int *vals, float *avg)
{
    // number of average calculations a single thread performs
    int avgCalcPerThread = ceilf(float(NUM_OF_AVG) / float(blockDim.x * gridDim.x));

    // thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // get first avg val for thread
    int avgIdx = idx * avgCalcPerThread;
    for (int i = 0; i < N && avgIdx < NUM_OF_AVG; ++i)
        avg[avgIdx] += vals[avgIdx + i];
    avg[avgIdx] /= N;

    // calculate the rest of avg vals for thread
    int maxAvgIdx = avgCalcPerThread * (idx + 1);
    for (avgIdx = idx * avgCalcPerThread + 1; 
         avgIdx < maxAvgIdx && avgIdx < NUM_OF_AVG; 
         ++avgIdx)
        avg[avgIdx] = (avg[avgIdx - 1] * N + vals[avgIdx + N - 1] - vals[avgIdx - 1]) / N;
}

int main()
{
    // ask user for grid and block dims, must multiply together to get NUM_OF_VALS
    cout << "Enter Grid X Dim: ";
    int gridDim; cin >> gridDim;

    cout << "Enter Block X Dim: ";
    int blockDim; cin >> blockDim;

    // create arrays of vals
    int vals[NUM_OF_VALS], *vals_d;
    float avg[NUM_OF_AVG], *avg_d;

    // create rng
	unsigned int seed = system_clock::now().time_since_epoch().count();
	std::default_random_engine generator(seed);
	std::uniform_int_distribution<int> dist(0, 5);

    // init vals
    for (int i = 0; i < NUM_OF_VALS; ++i)
    {
        vals[i] = dist(generator);

        if (i < NUM_OF_AVG)
            avg[i] = 0;
    }

    // memory sizes to allocate
    int valMem = sizeof(int) * NUM_OF_VALS, avgMem = sizeof(float) * NUM_OF_AVG;

    // allocate memory on device
    hipMalloc((void**)&vals_d, valMem);
    hipMalloc((void**)&avg_d, avgMem);
    
    // copy vals and avg to device
    hipMemcpy(vals_d, vals, valMem, hipMemcpyHostToDevice);
    hipMemcpy(avg_d, avg, avgMem, hipMemcpyHostToDevice);
    
    // call func
    movingAvg<<<gridDim, blockDim>>>(vals_d, avg_d);

    // copy device memory back to host
    hipMemcpy(avg, avg_d, avgMem, hipMemcpyDeviceToHost);

    /*
    // print vals
    for (int i = 0; i < NUM_OF_VALS; ++i)
        cout << vals[i] << ' ';
    cout << endl;

    // print averages
    for (int i = 0; i < NUM_OF_AVG; ++i)
        cout << avg[i] << ' ';
    cout << endl;
    */

    // free all device memory
    hipFree(vals_d); hipFree(avg_d); 
}